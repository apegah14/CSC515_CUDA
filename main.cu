#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <hip/driver_types.h>
#include ""

#define Tile_size 2

//Function To handle any errors occurred in the function calls


// Compute C = A * B
//*************************************************************
//Kernel for shared memory/ Tiled execution
__global__ void matrixMultiplyShared(float* A, float* B, float* C,
    int numARows, int numAColumns,
    int numBRows, int numBColumns,
    int numCRows, int numCColumns)
{
    __shared__ float sA[Tile_size][Tile_size];   // Tile size to store elements in shared memory
    __shared__ float sB[Tile_size][Tile_size];

    int Row = blockDim.y * blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1) / Tile_size) + 1); k++)
    {
        if ((Row < numARows) && (threadIdx.x + (k * Tile_size)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row * numAColumns) + threadIdx.x + (k * Tile_size)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (Col < numBColumns && (threadIdx.y + k * Tile_size) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * Tile_size) * numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < Tile_size; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
    {
        C[Row * numCColumns + Col] = Cvalue;
    }
}

__global__ void reluActivationForward(float* Z, float* A,
    int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < Z_x_dim * Z_y_dim) {
        A[index] = fmaxf(Z[index], 0);
    }
}

//*************************************************************
void Print_Mat(int Row, int Col, float* Mat)//Function To print the Matrix
{
    for (int i = 0; i < Row * Col; i++)
    {
        printf("%f  ", *(Mat + i));

        if ((i % Col) == 0)
        {
            printf("\n");
        }
    }
}//Function close
//*************************************************************
//Normal CPU Matrix Multiplication
void matMultiplyOnHost(float* A, float* B, float* C, int numARows,
    int numAColumns, int numBRows, int numBColumns,
    int numCRows, int numCColumns)
{
    for (int i = 0; i < numARows; i++)
    {
        for (int j = 0; j < numBColumns; j++)
        {
            C[i * numCColumns + j] = 0.0;
            for (int k = 0; k < numBRows; k++)
            {
                C[i * numCColumns + j] += A[i * numAColumns + k] * B[k * numBColumns + j];
            }
        }
    }
    return;
}
//*************************************************************
int input_rows = 1;
int input_cols = 784;
int layer1_rows = 784;
int layer1_cols = 128;
int layer2_rows = 128;
int layer2_cols = 10;
int output_rows = input_rows;
int output_cols = 10;

int main(int argc, char** argv) {
    float* host_input;
    float* host_output;

    float* host_layer1w;
    float* host_layer1b;
    float* host_layer1out;

    float* host_layer2w;
    float* host_layer2b;

    float* hostComputedC;

    float* device_input;
    float* device_output;

    float* device_layer1w;
    float* device_layer1b;
    float* device_layer1out_w;
    float* device_layer1out_b;

    float* device_layer2w;
    float* device_layer2b;

    // Please adjust rows and columns according to you need.

    //printf("\nPlease Enter Rows and Columns of A:");
    //scanf("%d %d", &numARows, &numAColumns);

    //printf("\nPlease Enter Rows and Columns of B:");
    //scanf("%d %d", &numBRows, &numBColumns);

    host_input = (float*)malloc(sizeof(float) * input_rows * input_cols);
    host_output = (float*)malloc(sizeof(float) * output_rows * output_cols);
    host_layer1w = (float*)malloc(sizeof(float) * layer1_rows * layer1_cols);
    host_layer1b = (float*)malloc(sizeof(float) * input_rows * layer1_cols);
    host_layer1out = (float*)malloc(sizeof(float) * input_rows * layer1_cols);
    host_layer2w = (float*)malloc(sizeof(float) * layer2_rows * layer2_cols);
    host_layer2b = (float*)malloc(sizeof(float) * input_rows * layer2_cols);
    hostComputedC = (float*)malloc(sizeof(float) * input_rows * layer1_cols);

    for (int i = 0; i < input_rows * input_cols; i++)//Matrix Initialization
    {
        host_input[i] = 1.0;
    }
    for (int i = 0; i < layer1_rows * layer1_cols; i++)
    {
        host_layer1w[i] = 1.0;
    }
    for (int i = 0; i < input_rows * layer1_cols; i++)
    {
        host_layer1b[i] = 1.0;
    }
    for (int i = 0; i < layer2_rows * layer2_cols; i++)
    {
        host_layer2w[i] = 1.0;
    }
    for (int i = 0; i < input_rows * layer2_cols; i++)
    {
        host_layer2b[i] = 1.0;
    }

    printf("\nMatrix A Values:\n");
    //Print_Mat(numARows, numAColumns, hostA);//Function Call

    printf("\n\nMatrix B Values:\n");
    //Print_Mat(numBRows, numBColumns, hostB);//Function Call

    // Allocating GPU memory
    hipMalloc((void**)&device_input, sizeof(float) * input_rows * input_cols);

    //Output vector
    hipMalloc((void**)&device_output, sizeof(float) * output_rows * output_cols);

    //Layer 1
    hipMalloc((void**)&device_layer1w, sizeof(float) * layer1_rows * layer1_cols);
    hipMalloc((void**)&device_layer1b, sizeof(float) * input_rows * layer1_cols);
    hipMalloc((void**)&device_layer1out_w, sizeof(float) * input_rows * layer1_cols);
    hipMalloc((void**)&device_layer1out_b, sizeof(float) * input_rows * layer1_cols);

    //Layer 2
    hipMalloc((void**)&device_layer2w, sizeof(float) * layer2_rows * layer2_cols);
    hipMalloc((void**)&device_layer2b, sizeof(float) * input_rows * layer2_cols);

    // Copy memory to the GPU
    hipMemcpy(device_input, host_input, sizeof(float) * input_rows * input_cols, hipMemcpyHostToDevice);
    hipMemcpy(device_output, host_output, sizeof(float) * output_rows * output_cols, hipMemcpyHostToDevice);
    hipMemcpy(device_layer1w, host_layer1w, sizeof(float) * layer1_rows * layer1_cols, hipMemcpyHostToDevice);
    hipMemcpy(device_layer1b, host_layer1b, sizeof(float) * input_rows * layer1_cols, hipMemcpyHostToDevice);
    hipMemcpy(device_layer2w, host_layer2w, sizeof(float) * layer2_rows * layer2_cols, hipMemcpyHostToDevice);
    hipMemcpy(device_layer2b, host_layer2b, sizeof(float) * input_rows * layer2_cols, hipMemcpyHostToDevice);

    // Initialize the grid and block dimensions

    dim3 dimGrid((layer1_cols / Tile_size) + 1, (input_rows / Tile_size) + 1, 1);//Number of Blocks required
    dim3 dimBlock(Tile_size, Tile_size, 1);//Number of threads in each block

    hipEvent_t start, stop;
    float time_w, time_b;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    //@@ Launch the GPU Kernel here
    hipEventRecord(start, 0);      // start time measurement
    matrixMultiplyShared << <dimGrid, dimBlock >> > (device_input, device_layer1w, device_layer1out_w, input_rows, input_cols, layer1_rows, layer1_cols, input_rows, layer1_cols);


    hipDeviceSynchronize();//To synchronize the device
    hipEventRecord(stop, 0);       // stop time measurement
    hipEventSynchronize(stop);     // sync results
    hipEventElapsedTime(&time_w, start, stop);
    printf("Elapsed time : %f ms\n", time_w);

    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

  

    // Copy the results in GPU memory back to the CPU
    hipMemcpy(host_layer1out, device_layer1out_w, sizeof(float) * input_rows * layer1_cols, hipMemcpyDeviceToHost);

    printf("\nMatrix C From Device\n");
    //Print_Mat(numCRows, numCColumns, hostC);//Function Call

    matMultiplyOnHost(host_input, host_layer1w, hostComputedC, input_rows, input_cols, layer1_rows, layer1_cols, input_rows, layer1_cols);

    printf("\nMatrix C From Host\n");
    //Print_Mat(numCRows, numCColumns, hostComputedC);//Function Call

    for (int i = 0; i < input_rows * layer1_cols; i++)//Compare both the result matrices 1. MatrixMultiplyonHost 2. MatrixMultiplyonDevice
    {
        if (hostComputedC[i] != host_layer1out[i])
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / layer1_cols, i % layer1_cols, hostComputedC[i], host_layer1out[i]);
            break;
        }
    }

    printf("\n Number of Blocks Created:%d \n", ((layer1_cols / Tile_size) + 1) * ((layer1_cols / Tile_size) + 1));
    printf("\n Number of Threads Per Block: %d \n", (Tile_size * Tile_size));
    //return 0;
    // Free the GPU memory
    free(host_input);
    free(host_output);
    free(host_layer1w);
    free(host_layer1b);
    free(host_layer1out);
    free(host_layer2w);
    free(host_layer2b);
    free(hostComputedC);

    hipFree(device_input);
    hipFree(device_output);

    hipFree(device_layer1w);
    hipFree(device_layer1b);
    hipFree(device_layer1out_w);
    hipFree(device_layer1out_b);

    hipFree(device_layer2w);
    hipFree(device_layer2b);
    int i = 1;
    //while (i > 0) {
        //printf("m");
    //}
    return 0;
}